#include "hip/hip_runtime.h"
#include "square.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>

__global__ void initRNG( hiprandState * rngStates, const unsigned int seed ) {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    hiprand_init( seed, idx, 0, &rngStates[idx] );
}

__global__ void square( Square::int_type * a, int N ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if( idx < N ) a[idx] = idx * idx;
}

__global__ void squareRNG( Square::int_type * a, int N, hiprandState * rngStates ) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState lState = rngStates[idx];

    Square::int_type r = hiprand(&lState);
    if( idx < N ) {
        a[idx] = r;
    }
}

Square::Square() : m_a(NULL), m_size(1), m_maxBlocks(1), m_maxThreadsPerBlock(256) {
    init();
}

void Square::init() {
    hipDeviceProp_t m_cdp;
    hipError_t err = hipGetDeviceProperties( &m_cdp, 0 );

    if( err != hipSuccess ) {
        std::cerr << "Unable to get device properties" << std::endl;
    } else {
        std::cerr << "Maximum Threads Per Block: " << m_cdp.maxThreadsPerBlock << std::endl;
        m_maxThreadsPerBlock = m_cdp.maxThreadsDim[0];
        m_maxBlocks = m_cdp.maxGridSize[0];
    }

//    std::cerr << "Maximum Threads Per Block: " << m_maxThreadsPerBlock << std::endl;
//    std::cerr << "Maximum Blocks: " << m_maxBlocks << std::endl;

    m_size = ((m_maxBlocks > 6 * m_maxThreadsPerBlock) ? 6 * m_maxThreadsPerBlock * m_maxThreadsPerBlock : m_maxBlocks * m_maxThreadsPerBlock);

    m_a = new int_type[ m_size ];

    size_t size = m_size * sizeof(int_type);
    hipMalloc( (void **) &m_dest, size);

//    std::cerr << "Compute Mode: " <<  m_cdp.computeMode << std::endl;
//    std::cerr << "Device Overlap: " <<  m_cdp.deviceOverlap << std::endl;
}

Square::~Square() {
    if( m_a ) delete [] m_a;
    hipFree( m_dest );
}

size_t Square::size() const { return m_size; }

void Square::operator()() {
    int block_count = (m_size / m_maxThreadsPerBlock );
    square<<<block_count,m_maxThreadsPerBlock>>>( m_dest, m_size );

    hipMemcpy( m_a, m_dest, m_size * sizeof(int_type), hipMemcpyDeviceToHost );
}

void Square::random_list() {
    int block_count = (m_size / m_maxThreadsPerBlock );
    hiprandState *d_rngStates = 0;
    hipMalloc( (void **) &d_rngStates, block_count * sizeof( d_rngStates ) );

    initRNG<<< 1, block_count >>>( d_rngStates, 1234 );

    squareRNG<<<1, block_count,m_maxThreadsPerBlock>>>( m_dest, m_size, d_rngStates );

    hipMemcpy( m_a, m_dest, m_size * sizeof(int_type), hipMemcpyDeviceToHost );
    hipFree( d_rngStates );
}

std::ostream & operator<<( std::ostream & out, const Square & rhs ) {
    for( unsigned int i = 0; i < rhs.size(); ++i ) {
        out << i << " -> " << rhs.m_a[i] << std::endl;
    }
    return out;
}
