#include "hip/hip_runtime.h"
//   Copyright 2015 Patrick Putnam
//
//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at
//
//       http://www.apache.org/licenses/LICENSE-2.0
//
//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.
#include "crossover_matrix.hpp"

#include "clotho/cuda/warp_sort.hpp"
//#include <iostream>

const unsigned int BLOCK_PER_ROW = 32;
const unsigned int ROW_PER_PAGE = 32;
const unsigned int MAX_THREADS = BLOCK_PER_ROW * ROW_PER_PAGE;

__shared__ double       g_sAlleles[ 1024 ];
__shared__ unsigned int g_sBuffer[ 1024 ];

__constant__ unsigned int g_cEvents[ crossover_wrapper::MAX_EVENTS + 1 ];

//inline std::ostream & operator<<( std::ostream & out, const dim3 & d ) {
//    out << "< " << d.x << ", " << d.y << ", " << d.z << " >";
//    return out;
//}

__global__ void generate_crossover_matrix4( double * rand_pool
                                            , double * alleles
                                            , unsigned int * sequences
                                            , dim3 max_dims ) {
    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

    unsigned int eStart = g_cEvents[blockIdx.y], eEnd = g_cEvents[blockIdx.y + 1];

    // if there are no recombination events for this thread block/sequence
    if( eStart >= eEnd ) {  // will be true or false for all threads in the block
        if( threadIdx.y == 0 ) {
            sequences[ (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.y ] = 0;
        }
        return;
    }

    __shared__ double recomb_points[ 1024 ];
    double rpt = 1.0;

    if( eStart + tid < eEnd ) {
        rpt = rand_pool[ eStart + tid ];
    }
    __syncthreads();

    // order the list of recombination points
    rpt = -log( rpt );

    // scan
    for( unsigned int i = 1; i < 32; i <<= 1 ) {
        double tmp = __shfl_up( rpt, i );
        if( threadIdx.x >= i ) rpt += tmp;
    }

    // share partial sums with other warps in block
    if( threadIdx.x == 31 ) {
        recomb_points[ threadIdx.y ] = rpt;
    }

    double accum = recomb_points[threadIdx.x];
    for( unsigned int i = 1; i < 32; i <<= 1 ) {
        double tmp = __shfl_up( accum, i );
        if( threadIdx.x >= i ) accum += tmp;
    }
    
    if( threadIdx.y > 0 ) {
        double tmp = __shfl( accum, threadIdx.y - 1 );
        rpt += tmp;
    }
    __syncthreads();

    double tmp = __shfl( accum, 31 );
    accum = tmp - log(0.1); // technically should be log of an additional random value (not a constant)

    rpt /= accum;
    recomb_points[tid] = rpt;
    // at this point rpt_{tid} are linearly ordered

    // load an allele into a register
    double allele = alleles[ blockIdx.x * 1024 + tid ];
    __syncthreads();

    // casting to int to allow for negative indices
    int min = 0, max = (eEnd - eStart);

    // binary search of recombination point list
    while( min <= max ) {
        unsigned int mid = ((max - min) / 2) + min;
        
        if( recomb_points[ mid ] < allele ) {
            min = mid + 1;
        } else if( recomb_points[ mid ] > allele ) {
            max = mid - 1;
        } else {
            // allele occurs at a recombination point
            min = mid;
            break;
        }
    }
    __syncthreads();
    // min contains count of preceeding recombination events

    unsigned int mask = ((min & 1) * (1 << threadIdx.x));

    for( unsigned int i = 1; i < 32; i <<= 1) {
        unsigned int m = __shfl_down(mask, i);
        if( !(threadIdx.x & ((i << 1) - 1)) ) mask |= m;
    }

    if(threadIdx.x == 0 ) {
        sequences[ (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.y ] = mask;
    }
}

__global__ void generate_crossover_matrix3( double * rand_pool
                                            , double * alleles
                                            , unsigned int * allele_mask
                                            , unsigned int * event_list
                                            , unsigned int * sequences
                                            , dim3 max_dims ) {
    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

    __shared__ unsigned int sEvents[ 1024 ];
    __shared__ double sOffset[ 1024 ];

    sEvents[ tid ] = event_list[blockIdx.y * 1024 + tid ];
    __syncthreads();

    unsigned int min_events = sEvents[0], max_events = sEvents[ 1023 ];
//    __syncthreads();

    // if there are no recombination events for this sequence
    if( min_events >= max_events ) {  // will be true or false for all threads in the block
        if( threadIdx.y == 0 ) {
            sequences[ blockIdx.y * max_dims.x + (blockIdx.x << 5) + threadIdx.x ] = 0;
        }
        return;
    }

    if( min_events + tid < max_events ) {
        sOffset[ tid ] = rand_pool[ min_events + tid ];
    }
    __syncthreads();    // used to allow for better memory access coalescing in subsequent step

    // load global memory into thread register(s)
    double all = alleles[ blockIdx.x * 1024 + tid ];
    unsigned int all_mask = allele_mask[ blockIdx.x * 1024 + tid ];
     __syncthreads();

    unsigned int idx = all * 1023;  // assumes all in [0,1.0)
    unsigned int lo_event_idx = sEvents[ idx ], hi_event_idx = sEvents[ idx + 1 ];

    lo_event_idx -= min_events;           // number of events before allele bin
    hi_event_idx -= min_events;

    // determine how many events occur **before** current allele in thi_event_idxs bin
    double rng_lo_event_idx = (double) idx / 1024.0, rng_hi_event_idx = ((double)(idx + 1))/ 1024.0;

    // expected to incur branch divergence
    while( lo_event_idx < hi_event_idx ) {
        double val = sOffset[lo_event_idx] * (rng_hi_event_idx - rng_lo_event_idx) + rng_lo_event_idx;
        if( val < all ) {
            // allele in right half of bin
            ++lo_event_idx;
            rng_lo_event_idx = val;
        } else {
            break;
        }
    }

    all_mask *= (lo_event_idx & 1);
    __syncthreads();    // necessary b/c of thread divergence

#pragma unroll
    for( unsigned int i = 2; i <= BLOCK_PER_ROW; i <<= 1 ) {
        unsigned int n = __shfl_down(all_mask, (i >> 1), BLOCK_PER_ROW );
        if( !(threadIdx.x & (i - 1))) all_mask |= n;
    }

    if( threadIdx.x == 0 ) {
        sequences[ blockIdx.y * max_dims.x + (blockIdx.x << 5) + threadIdx.y ] = all_mask;
    }
}

// kernel is invoked with each sequence as its own block Y coordinate
__global__ void generate_crossover_matrix2a( double * rand_pool
                                            , double * alleles
                                            , unsigned int * event_list
                                            , unsigned int * sequences
                                            , dim3 max_dims ) {
    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

    __shared__ double sRec[ 1024 ];

    unsigned int eStart = event_list[ max_dims.y +  blockIdx.y ], eEnd = event_list[ max_dims.y +  blockIdx.y + 1 ];
    __syncthreads();

    if( eStart >= eEnd ) {  // should be true or false for all threads in the block
        if( threadIdx.y == 0 ) {
            sequences[ blockIdx.y * max_dims.x + (blockIdx.x << 5) + threadIdx.x ] = 0;
        }
        __syncthreads();
        return;
    }

    double all = alleles[ blockIdx.x * 1024 + tid ];
    __syncthreads();

    unsigned int N = eEnd - eStart;
    if( tid < N ) {
        sRec[tid] = rand_pool[ eStart + tid ];
    }
    __syncthreads();

    double rng_lo = 0.0, rng_hi = 1.0;
    unsigned int prior_events = 0, event_idx = 0;
    while( N-- ) {
        double offset = sRec[ event_idx ];
        double r = rng_lo + offset * (rng_hi - rng_lo);

        unsigned int n = offset * N;
        if( r < all ) {
            // allele in right half of bin
            prior_events += n + 1;
            event_idx += n + 1;
            N -= n;
            rng_lo = r;
        } else {
            ++event_idx;
            N = n;
            rng_hi = r;
        }
    }
    __syncthreads();

    // if there are an odd number of preceeding crossover events
    // then this allele should be selected from the 'other' sequence
    // (or it exists in a crossover region)
    unsigned int m = ((prior_events & 1) * (1 << threadIdx.x));

#pragma unroll
    for( unsigned int i = 2; i <= BLOCK_PER_ROW; i <<= 1 ) {
        unsigned int n = __shfl_down(m, (i >> 1), BLOCK_PER_ROW );
        if( !(threadIdx.x & (i - 1))) m |= n;
    }

    // use one thread in each warp to write back to global memory
    // expecting this to not coalesce well
    // 
    if( threadIdx.x == 0 ) {
        sequences[ blockIdx.y * max_dims.x + (blockIdx.x << 5) + threadIdx.y ] = m;
    }
}

// kernel is invoked with each sequence as its own block Y coordinate
__global__ void generate_crossover_matrix2( double * rand_pool
                                            , double * alleles
                                            , unsigned int * event_list
                                            , unsigned int * sequences
                                            , dim3 max_dims ) {
    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;
//    unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;

//    __shared__ unsigned int sMask[ 32 ];
    __shared__ double sRec[ 1024 ];

    unsigned int eStart = event_list[ max_dims.y +  blockIdx.y ], eEnd = event_list[ max_dims.y +  blockIdx.y + 1 ];
    __syncthreads();

    if( eStart >= eEnd ) {  // should be true or false for all threads in the block
        if( threadIdx.y == 0 ) {
            sequences[ blockIdx.y * max_dims.x + (blockIdx.x << 5) + threadIdx.x ] = 0;
        }
        __syncthreads();
        return;
    }

    double all = alleles[ blockIdx.x * 1024 + tid ];
    __syncthreads();

    unsigned int mask = (1 << threadIdx.x);
    unsigned int m = 0;

    while( eStart < eEnd ) {
        __syncthreads();

        unsigned int N = (eEnd - eStart);
        if( N > 1024 ) { N = 1024; }

        sRec[ tid ] = (( tid < N ) ? rand_pool[ eStart + tid ] : -1.0);
        __syncthreads();

        eStart += N;

        while( N-- ) {
            double r = sRec[ N ];
            __syncthreads();
            m ^= (( 0 <= all && all < r) ? mask : 0 );
        }
    }
    __syncthreads();

#pragma unroll
    for( unsigned int i = 2; i <= BLOCK_PER_ROW; i <<= 1 ) {
        unsigned int n = __shfl_down(m, (i >> 1), BLOCK_PER_ROW );
        if( !(threadIdx.x & (i - 1))) m |= n;
    }
//    if( threadIdx.x == 0 ) {
//        sMask[ threadIdx.y ] = m;
//    }
//    __syncthreads();
//
//    if( threadIdx.y == 0 ) {
//        sequences[ blockIdx.y * max_dims.x + blockIdx.x * 32 + threadIdx.x ] = sMask[threadIdx.x];
//    }

    // use one thread in each warp to write back to global memory
    // expecting this to not coalesce well
    // 
    if( threadIdx.x == 0 ) {
        sequences[ blockIdx.y * max_dims.x + (blockIdx.x << 5) + threadIdx.y ] = m;
    }
}

__global__ void generate_crossover_matrix( double * rand_pool
                                         , double * alleles
                                         , unsigned int * event_list
                                         , unsigned int * sequences
                                         , dim3 max_dims ) {
    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

    // all threads within a block will operate upon a contiguous set of events and alleles
    __shared__ double lEvents[ MAX_THREADS ];       //  8K
    __shared__ unsigned int lMask[ 32 ];            //  4K
    __shared__ unsigned int lCounts[ MAX_THREADS ]; //  4K
                                                    // 24K shared memory

    // copy the alleles that this block is going to be inspecting
    unsigned int aidx = blockIdx.x * MAX_THREADS + tid;
    double all = ((aidx < max_dims.x ) ? alleles[aidx] : -1.0);
    __syncthreads();

    unsigned int seq_per_block = max_dims.y / gridDim.y;
    unsigned int unit_per_seq = max_dims.x / 32;

    unsigned int seq_start = blockIdx.y * seq_per_block, seq_end = seq_start + seq_per_block;

    if( seq_end > max_dims.y ) seq_end = max_dims.y;

    unsigned int c = 0, d = MAX_THREADS;
    unsigned int mask = (1 << threadIdx.x);

    while( seq_start < seq_end ) {
        if( d == MAX_THREADS ) {
            // refresh event count list
            c = seq_end - seq_start + 1;
            if( c > MAX_THREADS ) { c = MAX_THREADS; }

            lCounts[ tid ] = (( tid < c ) ? event_list[ seq_start + tid ] : 0);
            c = 0;
            d = 1;
        }
        __syncthreads();

        // number of events for sequence
        unsigned int eStart = lCounts[ c ], eEnd = lCounts[ d ];
        __syncthreads();
        c = d++;

        if( eStart < eEnd ) {
            unsigned int m = 0;
            do {
                // refresh event list

                // current method performs a 'lookahead copy'.
                // general question: is it better to perform a block copy from global memory to local memory,
                // or to simply access global memory as necessary?

                // Basic concern is that when N is small there will be a lot of sleeping threads
                unsigned int N = (eEnd - eStart);
                if( N > MAX_THREADS ) { N = MAX_THREADS; }
                lEvents[ tid ] = ((tid < N ) ? rand_pool[ eStart + tid ] : 0);
            
                __syncthreads();

                eStart += N;

                while( N-- ) {
                    double ev = lEvents[ N ];
                    __syncthreads();

                    m ^= (( 0 <= all && all < ev) ? mask : 0 );
                }
            } while( eStart < eEnd );
           __syncthreads();

#pragma unroll
            for( unsigned int i = 2; i <= BLOCK_PER_ROW; i <<= 1 ) {
                unsigned int n = __shfl_down(m, (i / 2), BLOCK_PER_ROW );
                if( !(threadIdx.x & (i - 1))) m |= n;
            }

            if( threadIdx.x == 0 ) {
                lMask[ threadIdx.y ] = m;
            }
        } else {
            if( threadIdx.y == 0 ) {
                lMask[ threadIdx.x ] = 0;
            }
        }
        __syncthreads();

        if( threadIdx.y == 0 ) { // threads [0, BLOCK_PER_ROW )
            // use a single warp to copy between shared and global memory
            sequences[ seq_start * unit_per_seq + tid ] =  lMask[threadIdx.x];
        }
        
        __syncthreads();

        ++seq_start;
    }
}

/*
__global__ void generate_crossover_matrix2( double * rand_pool, unsigned int * event_list, double * alleles, unsigned int * sequences, dim3 max_dims ) {
    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;
//    unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;
//
//    unsigned int tcount = blockDim.x * blockDim.y * blockDim.z;

    // all threads within a block will operate upon a contiguous set of events and alleles
    __shared__ double lEvent;
    __shared__ double lAlleles[ MAX_THREADS ];      //  8K
    __shared__ unsigned int lMask[ MAX_THREADS ];   //  4K
    __shared__ unsigned int lCounts[ MAX_THREADS ];  //  4K 
                                                    // 24K shared memory

    // copy the alleles that this block is going to be inspecting
    unsigned int aidx = blockIdx.x * MAX_THREADS + threadIdx.x;
    lAlleles[ tid ] = ((aidx < max_dims.x ) ? alleles[aidx] : -1.0);

    __syncthreads();

    unsigned int seq_per_block = max_dims.y / gridDim.y;

    unsigned int seq_start = blockIdx.y * seq_per_block, seq_end = seq_start + seq_per_block;

    if( seq_end > max_dims.y ) seq_end = max_dims.y;

    unsigned int c = 0;
    unsigned int mask = (1 << threadIdx.x);

    
    while( seq_start < seq_end ) {
        if( c == 0 ) {
            // refresh event count list
            c = seq_end - seq_start;
            if( c > MAX_THREADS ) { c = MAX_THREADS; }

            lCounts[ tid ] = (( tid < c ) ? event_list[ seq_start + tid ] : 0);
            --c;
        }
        __syncthreads();

        lMask[tid] = 0; // reset masks for sequence
        __syncthreads();

        // number of events for sequence
        unsigned int eEnd = lCounts[ c ], eStart = lCounts[ --c ];
        while( eStart < eEnd ) {
            if( tid == 0 ) {
                // use one thread to
                // get the next event from GLOBAL memory
                lEvent = rand_pool[ eStart ];
            }
            __syncthreads();

            lMask[tid] ^= (( lAlleles[tid] >= 0 && lAlleles[tid] < lEvent) ? mask : 0 );
            __syncthreads();
            ++eStart;
        }

        // collapse masks for sequence
        if( threadIdx.x & 1 == 0 ) {
            // half warp
            // merge odds and evens: 0 < 1; 2 < 3; 4 < 5 ...
            lMask[tid] |= lMask[tid + 1];
        }

        __syncthreads();

        if( threadIdx.x & 3 == 0 ) {
            // quarter warp
            // merge 0 < 2;4 < 6;8 < 10...
            lMask[tid] |= lMask[tid + 2];
        }

        __syncthreads();

        if( threadIdx.x & 7 == 0 ) {
            // eighth warp
            // merge 0 < 4; 8 < 12; 16 < 20; 24 < 28
            lMask[tid] |= lMask[tid + 4];
        }

        __syncthreads();

        if( threadIdx.x & 15 == 0 ) {
            // sixth warp
            // merge 0 < 8; 16 < 24
            lMask[tid] |= lMask[tid + 8];
        }

        __syncthreads();

        // copy sequence to output
        if( threadIdx.x & 31 == 0 ) {
            // thirty-second warp (1-thread)
            // merge 0 < 16
            lMask[ tid ] |= lMask[ tid + 16 ];
        }

        __syncthreads();

        if( tid < 32 ) {
            // use a single warp to copy between shared and global memory
            sequences[ seq_start * max_dims.x / 32 + tid ] = lMask[ tid * 32 ];
        }
        
        __syncthreads();
        ++seq_start;
    }
}*/

__global__ void crossover( unsigned int * seq
                            , double * alleles
                            , unsigned int allele_count
                            , double rpoint ) {
    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

    __shared__ unsigned int sSeq[32];

    // Thought: every thread block copies a relative
    // tile of the global allele and free_list
    // Technically these could exist in
    // shared memory as another kernel call with
    // the same blockIdx coordinates will copy
    // the same relative tile back into shared memory
    // ! Need to determine whether this repeated global access
    // is a significant overhead.
    // !! From an algorithmic perspective this may
    // be most general approach as it would avoid
    // scenarios where shared memory is exhausted
    unsigned int aidx = 32 * blockIdx.x * blockDim.x + tid;
    double sAllele = ((aidx < allele_count) ? alleles[aidx] : -1.0);
    __syncthreads();

    // load registers with global value
    unsigned int s = seq[ blockIdx.x * blockDim.x + threadIdx.x ];
    __syncthreads();

    unsigned int mask = (1 << threadIdx.x );
    unsigned int b = !!( (sAllele >= 0.0) && (sAllele < rpoint)); // rec point is after allele position
    s = ((s & mask) ^ ( b*mask));

    __syncthreads();

#pragma unroll
    for( unsigned int i = 2; i <= 32; i<<=1 ) {
        unsigned int tmp = __shfl_down( s, (i / 2), 32 );
        if( !(threadIdx.x & (i - 1)) ) s |= tmp;
    }

    if( threadIdx.x == 0 ) {
        sSeq[ threadIdx.y ] = s;
    }
    __syncthreads();

    if( threadIdx.y == 0 ) {
        seq[ blockIdx.x * blockDim.x + threadIdx.y ] = sSeq[threadIdx.x];
    }
}

__global__ void init_alleles( double * alleles, unsigned int count ) {
    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;
    g_sAlleles[ tid ] = ((tid < count ) ? alleles[ tid ] : -1.0);
}

__global__ void init_sequence( unsigned int * seq ) {
    unsigned int tid = threadIdx.y* blockDim.x + threadIdx.x;
    if( threadIdx.y == 0 ) {
        g_sBuffer[ threadIdx.x ] = seq[ threadIdx.x ];
    }
    __syncthreads();

    unsigned int mask = (1 << threadIdx.x );
    unsigned int v = g_sBuffer[ threadIdx.x ];
    __syncthreads();

    g_sBuffer[ tid ] = (v & mask);
}

__global__ void crossover2( double rpoint) {

    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

    double a = g_sAlleles[ tid ];
    unsigned int res = g_sBuffer[ tid ];
    __syncthreads();

    unsigned int mask = (1 << threadIdx.x);
    unsigned int b = !!((0.0 <= a) && (a < rpoint)); // rec point is after allele position
    res = (res ^ ( b*mask));

    g_sBuffer[tid] = res;
}

__global__ void finalize_sequence( unsigned int * seq ) {
    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

    unsigned int v = g_sBuffer[tid];
    __syncthreads();

#pragma unroll
    for( unsigned int i = 2; i <= 32; i<<=1 ) {
        unsigned int tmp = __shfl_down( v, (i / 2), 32 );
        if( !(threadIdx.x & (i - 1)) ) v |= tmp;
    }

    if( threadIdx.x == 0 ) {
        g_sBuffer[ threadIdx.y ] = v;
    }
    __syncthreads();

    if( threadIdx.y == 0 ) {
        seq[threadIdx.x] = g_sBuffer[threadIdx.x];
    }
}

void crossover_wrapper::operator()( double * rand_pool, double * alleles, unsigned int * event_list, unsigned int * sequences, dim3 max_dims ) {

    unsigned int seq_count = max_dims.y;
    while( seq_count ) {
        unsigned int N = seq_count;
        if( N > MAX_EVENTS ) { N = MAX_EVENTS; }
        hipMemcpyToSymbol(HIP_SYMBOL( event_list), g_cEvents, N + 1, hipMemcpyDeviceToDevice );

        dim3 blocks( max_dims.x / 1024, N, 1), threads( 32, 32, 1);

        generate_crossover_matrix4<<< blocks, threads >>>( rand_pool, alleles, sequences, max_dims );


//        std::cerr << blocks << ", " << threads << std::endl;
        hipDeviceSynchronize();

        event_list += N;
        seq_count -= N;
        sequences += (blocks.y * blocks.x * threads.x);
    }
}
