#include "hip/hip_runtime.h"
//   Copyright 2015 Patrick Putnam
//
//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at
//
//       http://www.apache.org/licenses/LICENSE-2.0
//
//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.
#include "crossover_matrix.hpp"

const unsigned int BLOCK_PER_ROW = 32;
const unsigned int ROW_PER_PAGE = 32;
const unsigned int MAX_THREADS = BLOCK_PER_ROW * ROW_PER_PAGE;

__shared__ double f_sAlleles[ 1024 ];
__shared__ unsigned int f_sSeq[ 1024 ];


// kernel is call each sequence has its own block
__global__ void generate_crossover_matrix2( double * rand_pool
                                            , double * alleles
                                            , unsigned int * event_list
                                            , unsigned int * sequences
                                            , dim3 max_dims ) {
    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;
//    unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;

//    __shared__ unsigned int sMask[ 32 ];
    __shared__ double sRec[ 1024 ];

    unsigned int eStart = event_list[ max_dims.y +  blockIdx.y ], eEnd = event_list[ max_dims.y +  blockIdx.y + 1 ];
    __syncthreads();

    if( eStart >= eEnd ) {  // should be true or false for all threads in the block
        if( threadIdx.y == 0 ) {
            sequences[ blockIdx.y * max_dims.x + blockIdx.x * 32 + threadIdx.x ] = 0;
        }
        __syncthreads();
        return;
    }

    double all = alleles[ blockIdx.x * 1024 + tid ];
    __syncthreads();

    unsigned int mask = (1 << threadIdx.x);
    unsigned int m = 0;

    while( eStart < eEnd ) {
        __syncthreads();

        unsigned int N = (eEnd - eStart);
        if( N > 1024 ) { N = 1024; }

        sRec[ tid ] = (( tid < N ) ? rand_pool[ eStart + tid ] : -1.0);
        __syncthreads();

        eStart += N;

        while( N-- ) {
            double r = sRec[ N ];
            __syncthreads();
            m ^= (( 0 <= all && all < r) ? mask : 0 );
        }
    }
    __syncthreads();

#pragma unroll
    for( unsigned int i = 2; i <= BLOCK_PER_ROW; i <<= 1 ) {
        unsigned int n = __shfl_down(m, (i / 2), BLOCK_PER_ROW );
        if( !(threadIdx.x & (i - 1))) m |= n;
    }
//    if( threadIdx.x == 0 ) {
//        sMask[ threadIdx.y ] = m;
//    }
//    __syncthreads();
//
//    if( threadIdx.y == 0 ) {
//        sequences[ blockIdx.y * max_dims.x + blockIdx.x * 32 + threadIdx.x ] = sMask[threadIdx.x];
//    }

    // use one thread in each warp to write back to global memory
    // expecting this to not coalesce well
    // 
    if( threadIdx.x == 0 ) {
        sequences[ blockIdx.y * max_dims.x + blockIdx.x * 32 + threadIdx.y ] = m;
    }
}

__global__ void generate_crossover_matrix( double * rand_pool
                                         , double * alleles
                                         , unsigned int * event_list
                                         , unsigned int * sequences
                                         , dim3 max_dims ) {
    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

    // all threads within a block will operate upon a contiguous set of events and alleles
    __shared__ double lEvents[ MAX_THREADS ];       //  8K
    __shared__ unsigned int lMask[ 32 ];            //  4K
    __shared__ unsigned int lCounts[ MAX_THREADS ]; //  4K
                                                    // 24K shared memory

    // copy the alleles that this block is going to be inspecting
    unsigned int aidx = blockIdx.x * MAX_THREADS + tid;
    double all = ((aidx < max_dims.x ) ? alleles[aidx] : -1.0);
    __syncthreads();

    unsigned int seq_per_block = max_dims.y / gridDim.y;
    unsigned int unit_per_seq = max_dims.x / 32;

    unsigned int seq_start = blockIdx.y * seq_per_block, seq_end = seq_start + seq_per_block;

    if( seq_end > max_dims.y ) seq_end = max_dims.y;

    unsigned int c = 0, d = MAX_THREADS;
    unsigned int mask = (1 << threadIdx.x);

    while( seq_start < seq_end ) {
        if( d == MAX_THREADS ) {
            // refresh event count list
            c = seq_end - seq_start + 1;
            if( c > MAX_THREADS ) { c = MAX_THREADS; }

            lCounts[ tid ] = (( tid < c ) ? event_list[ seq_start + tid ] : 0);
            c = 0;
            d = 1;
        }
        __syncthreads();

        // number of events for sequence
        unsigned int eStart = lCounts[ c ], eEnd = lCounts[ d ];
        __syncthreads();
        c = d++;

        if( eStart < eEnd ) {
            unsigned int m = 0;
            do {
                // refresh event list

                // current method performs a 'lookahead copy'.
                // general question: is it better to perform a block copy from global memory to local memory,
                // or to simply access global memory as necessary?

                // Basic concern is that when N is small there will be a lot of sleeping threads
                unsigned int N = (eEnd - eStart);
                if( N > MAX_THREADS ) { N = MAX_THREADS; }
                lEvents[ tid ] = ((tid < N ) ? rand_pool[ eStart + tid ] : 0);
            
                __syncthreads();

                eStart += N;

                while( N-- ) {
                    double ev = lEvents[ N ];
                    __syncthreads();

                    m ^= (( 0 <= all && all < ev) ? mask : 0 );
                }
            } while( eStart < eEnd );
           __syncthreads();

#pragma unroll
            for( unsigned int i = 2; i <= BLOCK_PER_ROW; i <<= 1 ) {
                unsigned int n = __shfl_down(m, (i / 2), BLOCK_PER_ROW );
                if( !(threadIdx.x & (i - 1))) m |= n;
            }

            if( threadIdx.x == 0 ) {
                lMask[ threadIdx.y ] = m;
            }
        } else {
            if( threadIdx.y == 0 ) {
                lMask[ threadIdx.x ] = 0;
            }
        }
        __syncthreads();

        if( threadIdx.y == 0 ) { // threads [0, BLOCK_PER_ROW )
            // use a single warp to copy between shared and global memory
            sequences[ seq_start * unit_per_seq + tid ] =  lMask[threadIdx.x];
        }
        
        __syncthreads();

        ++seq_start;
    }
}

/*
__global__ void generate_crossover_matrix2( double * rand_pool, unsigned int * event_list, double * alleles, unsigned int * sequences, dim3 max_dims ) {
    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;
//    unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;
//
//    unsigned int tcount = blockDim.x * blockDim.y * blockDim.z;

    // all threads within a block will operate upon a contiguous set of events and alleles
    __shared__ double lEvent;
    __shared__ double lAlleles[ MAX_THREADS ];      //  8K
    __shared__ unsigned int lMask[ MAX_THREADS ];   //  4K
    __shared__ unsigned int lCounts[ MAX_THREADS ];  //  4K 
                                                    // 24K shared memory

    // copy the alleles that this block is going to be inspecting
    unsigned int aidx = blockIdx.x * MAX_THREADS + threadIdx.x;
    lAlleles[ tid ] = ((aidx < max_dims.x ) ? alleles[aidx] : -1.0);

    __syncthreads();

    unsigned int seq_per_block = max_dims.y / gridDim.y;

    unsigned int seq_start = blockIdx.y * seq_per_block, seq_end = seq_start + seq_per_block;

    if( seq_end > max_dims.y ) seq_end = max_dims.y;

    unsigned int c = 0;
    unsigned int mask = (1 << threadIdx.x);

    
    while( seq_start < seq_end ) {
        if( c == 0 ) {
            // refresh event count list
            c = seq_end - seq_start;
            if( c > MAX_THREADS ) { c = MAX_THREADS; }

            lCounts[ tid ] = (( tid < c ) ? event_list[ seq_start + tid ] : 0);
            --c;
        }
        __syncthreads();

        lMask[tid] = 0; // reset masks for sequence
        __syncthreads();

        // number of events for sequence
        unsigned int eEnd = lCounts[ c ], eStart = lCounts[ --c ];
        while( eStart < eEnd ) {
            if( tid == 0 ) {
                // use one thread to
                // get the next event from GLOBAL memory
                lEvent = rand_pool[ eStart ];
            }
            __syncthreads();

            lMask[tid] ^= (( lAlleles[tid] >= 0 && lAlleles[tid] < lEvent) ? mask : 0 );
            __syncthreads();
            ++eStart;
        }

        // collapse masks for sequence
        if( threadIdx.x & 1 == 0 ) {
            // half warp
            // merge odds and evens: 0 < 1; 2 < 3; 4 < 5 ...
            lMask[tid] |= lMask[tid + 1];
        }

        __syncthreads();

        if( threadIdx.x & 3 == 0 ) {
            // quarter warp
            // merge 0 < 2;4 < 6;8 < 10...
            lMask[tid] |= lMask[tid + 2];
        }

        __syncthreads();

        if( threadIdx.x & 7 == 0 ) {
            // eighth warp
            // merge 0 < 4; 8 < 12; 16 < 20; 24 < 28
            lMask[tid] |= lMask[tid + 4];
        }

        __syncthreads();

        if( threadIdx.x & 15 == 0 ) {
            // sixth warp
            // merge 0 < 8; 16 < 24
            lMask[tid] |= lMask[tid + 8];
        }

        __syncthreads();

        // copy sequence to output
        if( threadIdx.x & 31 == 0 ) {
            // thirty-second warp (1-thread)
            // merge 0 < 16
            lMask[ tid ] |= lMask[ tid + 16 ];
        }

        __syncthreads();

        if( tid < 32 ) {
            // use a single warp to copy between shared and global memory
            sequences[ seq_start * max_dims.x / 32 + tid ] = lMask[ tid * 32 ];
        }
        
        __syncthreads();
        ++seq_start;
    }
}*/

__global__ void crossover( unsigned int * seq
                            , double * alleles
                            , unsigned int allele_count
                            , double rpoint ) {
    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

    __shared__ unsigned int sSeq[32];

    // Thought: every thread block copies a relative
    // tile of the global allele and free_list
    // Technically these could exist in
    // shared memory as another kernel call with
    // the same blockIdx coordinates will copy
    // the same relative tile back into shared memory
    // ! Need to determine whether this repeated global access
    // is a significant overhead.
    // !! From an algorithmic perspective this may
    // be most general approach as it would avoid
    // scenarios where shared memory is exhausted
    unsigned int aidx = 32 * blockIdx.x * blockDim.x + tid;
    double sAllele = ((aidx < allele_count) ? alleles[aidx] : -1.0);
    __syncthreads();

    // load registers with global value
    unsigned int s = seq[ blockIdx.x * blockDim.x + threadIdx.x ];
    __syncthreads();

    unsigned int mask = (1 << threadIdx.x );
    unsigned int b = !!( (sAllele >= 0.0) && (sAllele < rpoint)); // rec point is after allele position
    s = ((s & mask) ^ ( b*mask));

    __syncthreads();

#pragma unroll
    for( unsigned int i = 2; i <= 32; i<<=1 ) {
        unsigned int tmp = __shfl_down( s, (i / 2), 32 );
        if( !(threadIdx.x & (i - 1)) ) s |= tmp;
    }

    if( threadIdx.x == 0 ) {
        sSeq[ threadIdx.y ] = s;
    }
    __syncthreads();

    if( threadIdx.y == 0 ) {
        seq[ blockIdx.x * blockDim.x + threadIdx.y ] = sSeq[threadIdx.x];
    }
}

__global__ void init_alleles( double * alleles, unsigned int count ) {
    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;
    f_sAlleles[ tid ] = ((tid < count ) ? alleles[ tid ] : -1.0);
}

__global__ void init_sequence( unsigned int * seq ) {
    unsigned int tid = threadIdx.y* blockDim.x + threadIdx.x;
    if( threadIdx.y == 0 ) {
        f_sSeq[ threadIdx.x ] = seq[ threadIdx.x ];
    }
    __syncthreads();

    unsigned int mask = (1 << threadIdx.x );
    unsigned int v = f_sSeq[ threadIdx.x ];
    __syncthreads();

    f_sSeq[ tid ] = (v & mask);
}

__global__ void crossover2( double rpoint) {

    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

    double a = f_sAlleles[ tid ];
    unsigned int res = f_sSeq[ tid ];
    __syncthreads();

    unsigned int mask = (1 << threadIdx.x);
    unsigned int b = !!((0.0 <= a) && (a < rpoint)); // rec point is after allele position
    res = (res ^ ( b*mask));

    f_sSeq[tid] = res;
}

__global__ void finalize_sequence( unsigned int * seq ) {
    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

    unsigned int v = f_sSeq[tid];
    __syncthreads();

#pragma unroll
    for( unsigned int i = 2; i <= 32; i<<=1 ) {
        unsigned int tmp = __shfl_down( v, (i / 2), 32 );
        if( !(threadIdx.x & (i - 1)) ) v |= tmp;
    }

    if( threadIdx.x == 0 ) {
        f_sSeq[ threadIdx.y ] = v;
    }
    __syncthreads();

    if( threadIdx.y == 0 ) {
        seq[threadIdx.x] = f_sSeq[threadIdx.x];
    }
}
